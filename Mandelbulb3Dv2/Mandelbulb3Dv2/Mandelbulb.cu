#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Mandelbulb.cuh"
#include "Hypercomplex.cuh"
#include <GL/freeglut.h>
#include <GL/freeglut_ext.h>
#include <cmath>
#include <cstdio>
#include <ctime>
#define MAX(a, b) ((a) < (b) ? (b) : (a))
#define MIN(a, b) ((a) < (b) ? (b) : (a))
#define SIDE_MAX 1000

__device__ int side1;
__device__ int side2;
__device__ int side3;

__global__ void initVars(const int side)
{
	side1 = side;
	side2 = side * side;
	side3 = side * side * side;
}

__global__ void kernel(
	byte* buffer,
	const float n,
	const int maxIter,
	const float bailout,
	const float sqrBailout,
	int* counterPoints)
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	if (offset >= side3)
		return;
	int z = offset / side2;
	offset -= z * side2;
	int y = offset / side1;
	int x = offset % side1;
	offset += z * side2;

	// Compute point at this position
	int halfSide = side1 >> 1;
	float fx = bailout * (float)(x - halfSide) / halfSide;
	float fy = bailout * (float)(y - halfSide) / halfSide;
	float fz = bailout * (float)(z - halfSide) / halfSide;
	Hypercomplex hc(fx, fy, fz);
	Hypercomplex hz(fx, fy, fz);

	// Iterating
	bool belongs;
	if (hc.sqrRadius() > sqrBailout)
		belongs = false;
	else
	{
		for (int i = 0; i < maxIter; ++i)
			hz = (hz ^ n) + hc;
		belongs = hz.sqrRadius() <= sqrBailout;
	}

	if (belongs)
	{
		buffer[offset] = (byte)(hc.sqrRadius() / sqrBailout * 255);
		atomicAdd(counterPoints, 1);
	}
	else
		buffer[offset] = 0;
}

Mandelbulb::Mandelbulb(float power, int maxIter)
{
	this->n = power;
	this->maxIter = maxIter;
	this->bailout = powf(2.0f, 1.0f / (power - 1));
	this->sqrBailout = powf(4.0f, 1.0f / (power - 1));
}

bool Mandelbulb::compute(size_t width, size_t height)
{
	if (points)
		delete[] points;
	side = MIN(width, height);
	if (side > SIDE_MAX)
		side = SIDE_MAX;

	const size_t sz = side * side * side;
	points = new byte[sz];

	printf("Computing %d^3 points\n", side);
	// Processing
	byte* dev_buffer;

	if (hipMalloc((void**)&dev_buffer, sz) != hipSuccess)
	{
		printf("Error on creating buffer of pixels in GPU\n");
		return false;
	}

	clock_t tStart, tFinish;
	double tDelta;
	printf("Rendering %d^3\n", side);
	int threads = 1024;
	int blocks = (sz + threads - 1) / threads;
	int counterPoints = 0;
	int* dev_counterPoints;
	hipMalloc((void**)&dev_counterPoints, sizeof(int));
	hipMemcpy(dev_counterPoints, &counterPoints, sizeof(int), hipMemcpyHostToDevice);
	// Start
	tStart = clock();
	initVars << <1, 1 >> > (side);
	kernel << <blocks, threads >> > (dev_buffer, n, maxIter, bailout, sqrBailout, dev_counterPoints);
	hipDeviceSynchronize();
	tFinish = clock();
	// End
	hipMemcpy(&counterPoints, dev_counterPoints, sizeof(int), hipMemcpyDeviceToHost);
	tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("Included %d points (%.1f %%)\n",
		counterPoints,
		100.f * counterPoints / sz);
	printf("It tooks %.3f seconds\n", tDelta);

	if (hipMemcpy((void*)points, dev_buffer, sz, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error on getting buffer of pixels from GPU\n");
		return false;
	}
	hipFree(dev_buffer);
	hipFree(dev_counterPoints);

	printf("Cleaning of points\n");
	tStart = clock();
	int* pointsToCleaning = new int[sz];
	int cleaned = 0;
	int index = 0;
	for (int z = 1; z < side - 1; ++z)
	{
		for (int y = 1; y < side - 1; ++y)
		{
			for (int x = 1; x < side - 1; ++x)
			{
				int offset = z * side * side + y * side + x;
				if (points[offset] == 0)
					continue;
				else if (points[offset] == 1)
				{
					pointsToCleaning[index++] = offset;
					++cleaned;
					continue;
				}
				int offset000 = (z - 1) * side * side + (y - 1) * side + (x - 1);
				int offset001 = (z - 1) * side * side + (y - 1) * side + (x + 1);
				int offset010 = (z - 1) * side * side + (y + 1) * side + (x - 1);
				int offset011 = (z - 1) * side * side + (y + 1) * side + (x + 1);
				int offset100 = (z + 1) * side * side + (y - 1) * side + (x - 1);
				int offset101 = (z + 1) * side * side + (y - 1) * side + (x + 1);
				int offset110 = (z + 1) * side * side + (y + 1) * side + (x - 1);
				int offset111 = (z + 1) * side * side + (y + 1) * side + (x + 1);
				bool h000 = points[offset000] > 0;
				bool h001 = points[offset001] > 0;
				bool h010 = points[offset010] > 0;
				bool h011 = points[offset011] > 0;
				bool h100 = points[offset100] > 0;
				bool h101 = points[offset101] > 0;
				bool h110 = points[offset110] > 0;
				bool h111 = points[offset111] > 0;
				if (h000 && h001 && h010 && h011 && h100 && h101 && h110 && h111)
				{
					pointsToCleaning[index++] = offset;
					++cleaned;
				}
			}
		}
	}
	for (int i = 0; i < index; ++i)
		points[pointsToCleaning[i]] = 0;
	printf("Cleaned %d points (%.1f %%)\n",
		cleaned,
		100.f * cleaned / counterPoints);
	// End
	tFinish = clock();
	tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("It tooks %.3f seconds\n\n", tDelta);
	delete[] pointsToCleaning;
	return true;
}

void Mandelbulb::draw(size_t width, size_t height)
{
	if (points)
	{
		glBegin(GL_POINTS);
		int shiftX = (width - side) / 2 - width / 2;
		int shiftY = (height - side) / 2 - height / 2;
		int shiftZ = MAX(shiftX, shiftY);
		for (int z = 0; z < side; ++z)
		{
			for (int y = 0; y < side; ++y)
			{
				for (int x = 0; x < side; ++x)
				{
					int i = z * side * side + y * side + x;
					if (points[i] > 0)
					{
						int k = points[i];
						byte kRed = colorSpectrum[k][0];
						byte kGreen = colorSpectrum[k][1];
						byte kBlue = colorSpectrum[k][2];
						glColor3ub(
							kRed,
							kGreen,
							kBlue
						);
						glVertex3f(
							shiftX + x,
							shiftY + y,
							shiftZ + z
						);
					}
				}
			}
		}
		glEnd();
	}
}

//void Mandelbulb::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//		float b = 1 - 3 * k * (1 - k);
//
//		byte kRed = (byte)(4 * k * (1 - k) * 255);
//		byte kGreen = (byte)(k * 127);
//		byte kBlue = (byte)((1 - k) * 255);
//
//		colorSpectrum[i][0] = kRed * b;
//		colorSpectrum[i][1] = kGreen * b;
//		colorSpectrum[i][2] = kBlue * b;
//	}
//}

//void Mandelbulb::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//
//		byte kRed = (byte)(k * 255);
//		byte kGreen = (byte)(k * k * 255);
//		byte kBlue = (byte)((1 - 4 * k * (1 - k)) * 255);
//
//		colorSpectrum[i][0] = kRed;
//		colorSpectrum[i][1] = kGreen;
//		colorSpectrum[i][2] = kBlue;
//	}
//}

//void Mandelbulb::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//		float b = 1 - k * 0.3;
//
//		byte kRed = (byte)((k < 0.5 ? 2 * k : (k < 0.75 ? 1.0 : 3.25 - 3 * k)) * 255);
//		byte kGreen = (byte)((k < 0.5 ? 2 * k : (k < 0.75 ? 1.5 - k : 2.25 - 2 * k)) * 255);
//		byte kBlue = (byte)((k < 0.5 ? 1 : 2 - 2 * k) * 255);
//
//		colorSpectrum[i][0] = kRed * b;
//		colorSpectrum[i][1] = kGreen * b;
//		colorSpectrum[i][2] = kBlue * b;
//	}
//}

void Mandelbulb::initColorSpectrum()
{
	for (int i = 0; i < 256; ++i)
	{
		float k = i / 255.0;
		k = sqrtf(k);
		k = 4 * k * (1 - k);
		float b = 1 - 3 * k * (1 - k);

		byte kRed = (byte)((k < 0.5 ? 0 : (k < 0.75 ? 4 * k - 2 : 1.0)) * 255);
		byte kGreen = (byte)((k < 0.5 ? 0 : (k < 0.75 ? 2 * k - 1 : 0.5)) * 255);
		byte kBlue = (byte)((k < 0.5 ? 1 : 2 - 2 * k) * 255);

		colorSpectrum[i][0] = kRed * b;
		colorSpectrum[i][1] = kGreen * b;
		colorSpectrum[i][2] = kBlue * b;
	}
}