#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "QFractal.cuh"
#include "Quaternion.cuh"
#include <GL/freeglut.h>
#include <GL/freeglut_ext.h>
#include <cmath>
#include <cstdio>
#include <ctime>
#define MAX(a, b) ((a) < (b) ? (b) : (a))
#define MIN(a, b) ((a) < (b) ? (b) : (a))
#define SIDE_MAX 1000

__device__ int side1;
__device__ int side2;
__device__ int side3;

__global__ void initVars(const int side)
{
	side1 = side;
	side2 = side * side;
	side3 = side * side * side;
}

__global__ void kernel(
	byte* buffer,
	const float q1,
	const float q2,
	const float q3,
	const float q4,
	const int maxIter,
	const float bailout,
	const float sqrBailout,
	int* counterPoints)
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	if (offset >= side3)
		return;
	int z = offset / side2;
	offset -= z * side2;
	int y = offset / side1;
	int x = offset % side1;
	offset += z * side2;

	// Compute point at this position
	int halfSide = side1 >> 1;
	float fr = bailout * (float)(x - halfSide) / halfSide;
	float fa = bailout * (float)(y - halfSide) / halfSide;
	float fb = bailout * (float)(z - halfSide) / halfSide;
	float fc = q4;
	Quaternion qc(q1, q2, q3, q4);
	Quaternion qv(fr, fa, fb, fc);

	// Iterating
	bool belongs;
	if (qv.sqrRadius() > sqrBailout)
		belongs = false;
	else
	{
		for (int i = 0; i < maxIter; ++i)
			qv = qv.sqr() + qc;
		belongs = qv.sqrRadius() <= sqrBailout;
	}

	if (belongs)
	{
		buffer[offset] = (byte)((fr * fr + fa * fa + fb * fb) / (sqrBailout - fc * fc) * 255);
		atomicAdd(counterPoints, 1);
	}
	else
		buffer[offset] = 0;
}

QFractal::QFractal(float r, float a, float b, float c, QFractal::ParamToHide h, int maxIter)
{
	switch (h)
	{
	case QFractal::R:
		this->q1 = a;
		this->q2 = b;
		this->q3 = c;
		this->q4 = r;
		break;
	case QFractal::A:
		this->q1 = r;
		this->q2 = b;
		this->q3 = c;
		this->q4 = a;
		break;
	case QFractal::B:
		this->q1 = r;
		this->q2 = a;
		this->q3 = c;
		this->q4 = b;
		break;
	case QFractal::C:
		this->q1 = r;
		this->q2 = a;
		this->q3 = b;
		this->q4 = c;
		break;
	}
	this->maxIter = maxIter;
	this->bailout = 2.0f;
	this->sqrBailout = 4.0f;
}

bool QFractal::compute(size_t width, size_t height)
{
	if (points)
		delete[] points;
	side = MIN(width, height);
	if (side > SIDE_MAX)
		side = SIDE_MAX;

	const size_t sz = side * side * side;
	points = new byte[sz];

	printf("Computing %d^3 points\n", side);
	// Processing
	byte* dev_buffer;

	if (hipMalloc((void**)&dev_buffer, sz) != hipSuccess)
	{
		printf("Error on creating buffer of pixels in GPU\n");
		return false;
	}

	clock_t tStart, tFinish;
	double tDelta;
	printf("Rendering %d^3\n", side);
	int threads = 1024;
	int blocks = (sz + threads - 1) / threads;
	int counterPoints = 0;
	int* dev_counterPoints;
	hipMalloc((void**)&dev_counterPoints, sizeof(int));
	hipMemcpy(dev_counterPoints, &counterPoints, sizeof(int), hipMemcpyHostToDevice);
	// Start
	tStart = clock();
	initVars << <1, 1 >> > (side);
	kernel << <blocks, threads >> > (dev_buffer, q1, q2, q3, q4, maxIter, bailout, sqrBailout, dev_counterPoints);
	hipDeviceSynchronize();
	tFinish = clock();
	// End
	hipMemcpy(&counterPoints, dev_counterPoints, sizeof(int), hipMemcpyDeviceToHost);
	tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("Included %d points (%.1f %%)\n",
		counterPoints,
		100.f * counterPoints / sz);
	printf("It tooks %.3f seconds\n", tDelta);

	if (hipMemcpy((void*)points, dev_buffer, sz, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error on getting buffer of pixels from GPU\n");
		return false;
	}
	hipFree(dev_buffer);
	hipFree(dev_counterPoints);

	printf("Cleaning of points\n");
	tStart = clock();
	int* pointsToCleaning = new int[sz];
	int cleaned = 0;
	int index = 0;
	for (int z = 1; z < side - 1; ++z)
	{
		for (int y = 1; y < side - 1; ++y)
		{
			for (int x = 1; x < side - 1; ++x)
			{
				int offset = z * side * side + y * side + x;
				if (points[offset] == 0)
					continue;
				else if (points[offset] == 1)
				{
					pointsToCleaning[index++] = offset;
					++cleaned;
					continue;
				}
				int offset000 = (z - 1) * side * side + (y - 1) * side + (x - 1);
				int offset001 = (z - 1) * side * side + (y - 1) * side + (x + 1);
				int offset010 = (z - 1) * side * side + (y + 1) * side + (x - 1);
				int offset011 = (z - 1) * side * side + (y + 1) * side + (x + 1);
				int offset100 = (z + 1) * side * side + (y - 1) * side + (x - 1);
				int offset101 = (z + 1) * side * side + (y - 1) * side + (x + 1);
				int offset110 = (z + 1) * side * side + (y + 1) * side + (x - 1);
				int offset111 = (z + 1) * side * side + (y + 1) * side + (x + 1);
				bool h000 = points[offset000] > 0;
				bool h001 = points[offset001] > 0;
				bool h010 = points[offset010] > 0;
				bool h011 = points[offset011] > 0;
				bool h100 = points[offset100] > 0;
				bool h101 = points[offset101] > 0;
				bool h110 = points[offset110] > 0;
				bool h111 = points[offset111] > 0;
				if (h000 && h001 && h010 && h011 && h100 && h101 && h110 && h111)
				{
					pointsToCleaning[index++] = offset;
					++cleaned;
					continue;
				}
			}
		}
	}
	for (int i = 0; i < index; ++i)
		points[pointsToCleaning[i]] = 0;
	printf("Cleaned %d points (%.1f %%)\n",
		cleaned,
		100.f * cleaned / counterPoints);
	// End
	tFinish = clock();
	tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("It tooks %.3f seconds\n\n", tDelta);
	delete[] pointsToCleaning;
	return true;
}

void QFractal::draw(size_t width, size_t height)
{
	if (points)
	{
		glBegin(GL_POINTS);
		int shiftX = (width - side) / 2 - width / 2;
		int shiftY = (height - side) / 2 - height / 2;
		int shiftZ = MAX(shiftX, shiftY);
		for (int z = 0; z < side; ++z)
		{
			for (int y = 0; y < side; ++y)
			{
				for (int x = 0; x < side; ++x)
				{
					int i = z * side * side + y * side + x;
					if (points[i] > 0)
					{
						int k = points[i];
						byte kRed = colorSpectrum[k][0];
						byte kGreen = colorSpectrum[k][1];
						byte kBlue = colorSpectrum[k][2];
						glColor3ub(
							kRed,
							kGreen,
							kBlue
						);
						glVertex3f(
							shiftX + x,
							shiftY + y,
							shiftZ + z
						);
					}
				}
			}
		}
		glEnd();
	}
}

//void QFractal::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = 1.0 * i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//		float b = 1 - 3 * k * (1 - k);
//
//		byte kRed = (byte)(4 * k * (1 - k) * 255);
//		byte kGreen = (byte)(k * 127);
//		byte kBlue = (byte)((1 - k) * 255);
//
//		colorSpectrum[i][0] = kRed * b;
//		colorSpectrum[i][1] = kGreen * b;
//		colorSpectrum[i][2] = kBlue * b;
//	}
//}

//void QFractal::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//
//		byte kRed = (byte)(k * 255);
//		byte kGreen = (byte)(k * k * 255);
//		byte kBlue = (byte)((1 - 4 * k * (1 - k)) * 255);
//
//		colorSpectrum[i][0] = kRed;
//		colorSpectrum[i][1] = kGreen;
//		colorSpectrum[i][2] = kBlue;
//	}
//}

//void QFractal::initColorSpectrum()
//{
//	for (int i = 0; i < 256; ++i)
//	{
//		float k = i / 255.0;
//		k = sqrtf(k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//		k = 4 * k * (1 - k);
//		float b = 2 * k * (1 - k) + 0.5;
//
//		byte kRed = (byte)((k < 0.5 ? 2 * k : (k < 0.75 ? 1.0 : 3.25 - 3 * k)) * 255);
//		byte kGreen = (byte)((k < 0.5 ? 2 * k : (k < 0.75 ? 1.5 - k : 2.25 - 2 * k)) * 255);
//		byte kBlue = (byte)((k < 0.5 ? 1 : 2 - 2 * k) * 255);
//
//		colorSpectrum[i][0] = kRed * b;
//		colorSpectrum[i][1] = kGreen * b;
//		colorSpectrum[i][2] = kBlue * b;
//	}
//}

void QFractal::initColorSpectrum()
{
	for (int i = 0; i < 256; ++i)
	{
		float k = i / 255.0;
		k = sqrtf(k);
		k = 4 * k * (1 - k);
		k = 4 * k * (1 - k);
		k = 4 * k * (1 - k);
		float b = 1 - 3 * k * (1 - k);

		byte kRed = (byte)((k < 0.5 ? 0 : (k < 0.75 ? 4 * k - 2 : 1.0)) * 255);
		byte kGreen = (byte)((k < 0.5 ? 0 : (k < 0.75 ? 2 * k - 1 : 0.5)) * 255);
		byte kBlue = (byte)((k < 0.5 ? 1 : 2 - 2 * k) * 255);

		colorSpectrum[i][0] = kRed * b;
		colorSpectrum[i][1] = kGreen * b;
		colorSpectrum[i][2] = kBlue * b;
	}
}
